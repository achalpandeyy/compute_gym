#include "hip/hip_runtime.h"
#include "core_types.h"
#include "core_memory.h"

#include "common.cuh"

template <typename T>
static void SequentialInclusiveScan(int count, T *array)
{
    for (int i = 1; i < count; ++i)
    {
        array[i] += array[i - 1];
    }
}

template <typename T>
static void SequentialExclusiveScan(int count, T *array)
{
    T accum = T(0);
    for (int i = 0; i < count; ++i)
    {
        T temp = accum;
        accum += array[i];
        array[i] = temp;
    }
}

#define SEGMENT_SIZE 1024

template <typename T>
__device__ T BlockScan1(T *block)
{   
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        __syncthreads();

        T temp = T(0);
        if (threadIdx.x >= stride)
            temp = block[threadIdx.x - stride];
        __syncthreads();

        block[threadIdx.x] = block[threadIdx.x] + temp;
    }
    
    return block[threadIdx.x];
}

template <typename T>
__global__ void InclusiveScanUpsweep(u64 count, T *array, T *summary)
{
    __shared__ T segment[SEGMENT_SIZE];

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < count)
        segment[threadIdx.x] = array[index];
    else
        segment[threadIdx.x] = T(0);
    __syncthreads();

    T scan_result = BlockScan1(segment);

    if (index < count)
        array[index] = scan_result;

    if (summary && (threadIdx.x == blockDim.x - 1))
        summary[blockIdx.x] = scan_result;
}

template <typename T>
__global__ void InclusiveScanDownsweep(u64 count, T *array, T *summary)
{
    if (blockIdx.x > 0)
    {
        int prev_block_index = blockIdx.x - 1;
        T prev_block_sum = summary[prev_block_index];
        
        int index = blockIdx.x*blockDim.x + threadIdx.x;
        if (index < count)
            array[index] = array[index] + prev_block_sum;
    }
}

template <typename T>
__global__ void ExclusiveScanUpsweep(u64 count, T *array, T *summary)
{
    __shared__ T segment[SEGMENT_SIZE];

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < count && threadIdx.x != 0)
        segment[threadIdx.x] = array[index - 1];
    else
        segment[threadIdx.x] = T(0);

    T scan_result = BlockScan1(segment);

    if (summary && (threadIdx.x == blockDim.x - 1))
        summary[blockIdx.x] = scan_result + array[index];

    if (index < count)
        array[index] = scan_result;
}

template <typename T>
__global__ void ExclusiveScanDownsweep(u64 count, T *array, T *summary)
{
    if (blockIdx.x > 0)
    {
        int block_index = blockIdx.x;
        T block_sum = summary[block_index];
        
        int index = blockIdx.x*blockDim.x + threadIdx.x;
        if (index < count)
            array[index] = array[index] + block_sum;
    }
}

template <typename T>
struct Scan_PassInput
{
    T *d_array;
    T *d_summary;
    u64 element_count;
};

template <typename T>
struct Scan_Input
{
    T *d_scratch;
    u32 pass_input_count;
    Scan_PassInput<T> *pass_inputs;
};

template <typename T>
static Scan_Input<T> *Scan_CreateInput(Arena *arena, u64 count, T *d_array, int block_dim)
{
    Scan_Input<T> *input = PushStruct(arena, Scan_Input<T>);

    // Allocate scratch
    {
        u64 size = 0ull;

        // Add the number of elements for the first pass..
        u64 out_count = (count + block_dim - 1)/block_dim;
        size += out_count*sizeof(*input->d_scratch);

        // Add the number of elements for the second pass.. and we are done because we can just ping-pong.
        out_count = (out_count + block_dim - 1)/block_dim;
        size += out_count*sizeof(*input->d_scratch);

        CUDACheck(hipMalloc(&input->d_scratch, size));
    }

    input->pass_input_count = (u32)ceilf(logf(count)/logf(block_dim));
    input->pass_inputs = PushArrayZero(arena, Scan_PassInput<T>, input->pass_input_count);

    int out_count = (count + block_dim - 1)/block_dim; // output of the first pass
    T *d_pingpong[] = {input->d_scratch, input->d_scratch + out_count};
    u64 element_count = count;

    for (u32 i = 0; i < input->pass_input_count; ++i)
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + i;
        pass_input->d_array = d_array;
        pass_input->d_summary = d_pingpong[i % 2];
        pass_input->element_count = element_count;

        int grid_dim = (element_count + block_dim - 1)/block_dim;
        
        d_array = pass_input->d_summary;
        element_count = grid_dim;
    }

    input->pass_inputs[input->pass_input_count - 1].d_summary = 0; // the "top" pass doesn't require a summary

    return input;
}

template <typename T>
static void Scan_DestroyInput(Scan_Input<T> *input)
{
    CUDACheck(hipFree(input->d_scratch));
}

template <typename T>
static void InclusiveScan1(Scan_Input<T> *input, hipStream_t stream)
{
    int block_dim = SEGMENT_SIZE;
    u32 k = input->pass_input_count;

    for (u32 pass = 0; pass < k; ++pass) // k upsweeps
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + pass;
        int grid_dim = (pass_input->element_count + block_dim - 1)/block_dim;
        CUDACheck((InclusiveScanUpsweep<T><<<grid_dim, block_dim, 0, stream>>>(pass_input->element_count, pass_input->d_array, pass_input->d_summary)));
    }

    for (s32 pass = k - 2; pass >= 0; --pass) // k - 1 downsweeps
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + pass;
        int grid_dim = (pass_input->element_count + block_dim - 1)/block_dim;
        CUDACheck((InclusiveScanDownsweep<T><<<grid_dim, block_dim, 0, stream>>>(pass_input->element_count, pass_input->d_array, pass_input->d_summary)));
    }
}

template <typename T>
static void ExclusiveScan1(Scan_Input<T> *input, hipStream_t stream)
{
    int block_dim = SEGMENT_SIZE;
    u32 k = input->pass_input_count;

    for (u32 pass = 0; pass < k; ++pass) // k upsweeps
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + pass;
        int grid_dim = (pass_input->element_count + block_dim - 1)/block_dim;
        CUDACheck((ExclusiveScanUpsweep<T><<<grid_dim, block_dim, 0, stream>>>(pass_input->element_count, pass_input->d_array, pass_input->d_summary)));
    }

    for (s32 pass = k - 2; pass >= 0; --pass) // k - 1 downsweeps
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + pass;
        int grid_dim = (pass_input->element_count + block_dim - 1)/block_dim;
        CUDACheck((ExclusiveScanDownsweep<T><<<grid_dim, block_dim, 0, stream>>>(pass_input->element_count, pass_input->d_array, pass_input->d_summary)));
    }
}

#include "benchmarking.cu"

#include <thrust/scan.h>

using InputType = int;

template <typename T>
void ThrustInclusiveScan(u64 count, T *input, T *output, hipStream_t stream)
{
    thrust::inclusive_scan(thrust::cuda::par.on(stream), input, input + count, output);
}

template <typename T>
void ThrustExclusiveScan(u64 count, T *input, T *output, hipStream_t stream)
{
    thrust::exclusive_scan(thrust::cuda::par.on(stream), input, input + count, output);
}

template <typename T>
struct Data
{
    u64 count;
    T *h_array;
    T *d_array;
    Scan_Input<T> *input;
};

template <typename T>
static Data<T> *CreateData(Arena *arena, u64 count, hipStream_t stream)
{
    Data<T> *data = PushStruct(arena, Data<T>);
    data->count = count;

    data->h_array = PushArray(arena, T, data->count);
    for (u64 i = 0; i < data->count; ++i)
        data->h_array[i] = T(1);

    CUDACheck(hipMalloc(&data->d_array, data->count*sizeof(*data->d_array)));
    CUDACheck(hipMemcpyAsync(data->d_array, data->h_array, data->count*sizeof(*data->d_array), hipMemcpyHostToDevice, stream));

    CUDACheck(hipStreamSynchronize(stream));

    data->input = Scan_CreateInput(arena, data->count, data->d_array, SEGMENT_SIZE);

    return data;
}

template <typename T>
static void DestroyData(Data<T> *data)
{
    Scan_DestroyInput(data->input);
    CUDACheck(hipFree(data->d_array));
}

template <typename T>
static b32 ValidateGPUOutput(Arena *arena, Data<T> *data)
{
    Scratch scratch = ScratchBegin(arena);

    T *gpu_out = PushArray(scratch.arena, T, data->count);
    CUDACheck(hipMemcpy(gpu_out, data->d_array, data->count*sizeof(*gpu_out), hipMemcpyDeviceToHost));

    SequentialInclusiveScan<T>(data->count, data->h_array);
    b32 result = (memcmp(gpu_out, data->h_array, data->count*sizeof(*gpu_out)) == 0);

    ScratchEnd(&scratch);

    return result;
}

template <typename T>
static void FunctionToBenchmark(Data<T> *data, hipStream_t stream)
{
    InclusiveScan1<T>(data->input, stream);
    // ThrustInclusiveScan<T>(data->count, data->d_array, data->d_array, stream);
}

static void TestScan();

int main(int argc, char **argv)
{
    char *file_name = 0;
    if (argc == 2)
    {
        file_name = argv[1];
    }

    if (1)
    {
        TestScan();
        printf("All tests passed\n");
    }

    f64 peak_gbps = 0.0;
    f64 peak_gflops = 0.0;
    GetPeakMeasurements(&peak_gbps, &peak_gflops, true);
    
    printf("Bandwidth (Peak):\t%.2f GBPS\n", peak_gbps);
    printf("Throughput (Peak):\t%.2f GFLOPS\n", peak_gflops);
    printf("Arithmetic intensity (Peak):\t%.2f FLOPS/byte\n", peak_gflops/peak_gbps);

    Benchmark<InputType>(peak_gbps, peak_gflops, file_name);

    return 0;
}

static void TestScan()
{
    srand(0);

    int test_count = 10;
    while (test_count > 0)
    {
        
        u64 count = GetRandomNumber(30);
        printf("Count: %llu:\t", count);
        
        // Inclusive scan
        {
            printf("Inclusive, ");
            Scratch scratch = ScratchBegin(GetScratchArena(GigaBytes(10)));
            
            Data<int> *data = CreateData<int>(scratch.arena, count, 0);
            Scan_Input<int> *input = Scan_CreateInput(scratch.arena, count, data->d_array, SEGMENT_SIZE);
            InclusiveScan1<int>(input, 0);
            
            int *gpu_output = PushArray(scratch.arena, int, count);
            CUDACheck(hipMemcpy(gpu_output, data->d_array, count*sizeof(int), hipMemcpyDeviceToHost));

            SequentialInclusiveScan<int>(count, data->h_array);
            for (int i = 0; i < count; ++i)
            {
                if (data->h_array[i] != gpu_output[i])
                {
                    printf("[FAIL] Result[%d] (GPU): %d \tExpected: %d\n", i, gpu_output[i], data->h_array[i]);
                    exit(1);
                }
            }
            printf("Passed\t");

            DestroyData<int>(data);

            ScratchEnd(&scratch);
        }

        // Exclusive scan
        {
            printf("Exclusive, ");
            Scratch scratch = ScratchBegin(GetScratchArena(GigaBytes(10)));

            Data<int> *data = CreateData<int>(scratch.arena, count, 0);
            Scan_Input<int> *input = Scan_CreateInput(scratch.arena, count, data->d_array, SEGMENT_SIZE);
            ExclusiveScan1<int>(input, 0);
            
            int *gpu_output = PushArray(scratch.arena, int, count);
            CUDACheck(hipMemcpy(gpu_output, data->d_array, count*sizeof(int), hipMemcpyDeviceToHost));

            SequentialExclusiveScan<int>(count, data->h_array);
            for (int i = 0; i < count; ++i)
            {
                if (data->h_array[i] != gpu_output[i])
                {
                    printf("[FAIL] Result[%d] (GPU): %d \tExpected: %d\n", i, gpu_output[i], data->h_array[i]);
                    exit(1);
                }
            }
            printf("Passed\n");

            DestroyData<int>(data);
            ScratchEnd(&scratch);
        }

        --test_count;
    }
}