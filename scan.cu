#include "hip/hip_runtime.h"
#include "core_types.h"
#include "core_memory.h"

#include "common.cuh"

template <typename T>
static void SequentialScan(int count, T *array)
{
    for (int i = 1; i < count; ++i)
    {
        array[i] += array[i - 1];
    }
}

#define SEGMENT_SIZE 1024

template <typename T>
__device__ T BlockScan1(T *block)
{   
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        __syncthreads();

        T temp = T(0);
        if (threadIdx.x >= stride)
            temp = block[threadIdx.x - stride];
        __syncthreads();

        block[threadIdx.x] = block[threadIdx.x] + temp;
    }
    
    return block[threadIdx.x];
}

template <typename T>
__global__ void ScanUpsweep(u64 count, T *array, T *summary)
{
    __shared__ T segment[SEGMENT_SIZE];

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < count)
        segment[threadIdx.x] = array[index];
    else
        segment[threadIdx.x] = T(0);

    T scan_result = BlockScan1(segment);

    if (index < count)
        array[index] = scan_result;

    if (summary && (threadIdx.x == blockDim.x - 1))
        summary[blockIdx.x] = scan_result;
}

template <typename T>
__global__ void ScanDownsweep(u64 count, T *array, T *summary)
{
    if (blockIdx.x > 0)
    {
        int prev_block_index = blockIdx.x - 1;
        T prev_block_sum = summary[prev_block_index];
        
        int index = blockIdx.x*blockDim.x + threadIdx.x;
        if (index < count)
            array[index] += prev_block_sum;
    }
}

template <typename T>
struct Scan_PassInput
{
    T *d_array;
    T *d_summary;
    u64 element_count;
};

template <typename T>
struct Scan_Input
{
    T *d_scratch;
    u32 pass_input_count;
    Scan_PassInput<T> *pass_inputs;
};

template <typename T>
static Scan_Input<T> *Scan_CreateInput(Arena *arena, u64 count, T *d_array, int block_dim)
{
    Scan_Input<T> *input = PushStruct(arena, Scan_Input<T>);

    // Allocate scratch
    {
        u64 size = 0ull;

        // Add the number of elements for the first pass..
        u64 out_count = (count + block_dim - 1)/block_dim;
        size += out_count*sizeof(*input->d_scratch);

        // Add the number of elements for the second pass.. and we are done because we can just ping-pong.
        out_count = (out_count + block_dim - 1)/block_dim;
        size += out_count*sizeof(*input->d_scratch);

        CUDACheck(hipMalloc(&input->d_scratch, size));
    }

    input->pass_input_count = (u32)ceilf(logf(count)/logf(block_dim));
    input->pass_inputs = PushArrayZero(arena, Scan_PassInput<T>, input->pass_input_count);

    int out_count = (count + block_dim - 1)/block_dim; // output of the first pass
    T *d_pingpong[] = {input->d_scratch, input->d_scratch + out_count};
    u64 element_count = count;

    for (u32 i = 0; i < input->pass_input_count; ++i)
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + i;
        pass_input->d_array = d_array;
        pass_input->d_summary = d_pingpong[i % 2];
        pass_input->element_count = element_count;

        int grid_dim = (element_count + block_dim - 1)/block_dim;
        
        d_array = pass_input->d_summary;
        element_count = grid_dim;
    }

    input->pass_inputs[input->pass_input_count - 1].d_summary = 0; // the "top" pass doesn't require a summary

    return input;
}

template <typename T>
static void Scan_DestroyInput(Scan_Input<T> *input)
{
    CUDACheck(hipFree(input->d_scratch));
}

template <typename T>
static void Scan1(Scan_Input<T> *input, hipStream_t stream)
{
    int block_dim = SEGMENT_SIZE;
    u32 k = input->pass_input_count;

    for (u32 pass = 0; pass < k; ++pass) // k upsweeps
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + pass;
        int grid_dim = (pass_input->element_count + block_dim - 1)/block_dim;
        CUDACheck((ScanUpsweep<T><<<grid_dim, block_dim, 0, stream>>>(pass_input->element_count, pass_input->d_array, pass_input->d_summary)));
    }

    for (s32 pass = k - 2; pass >= 0; --pass) // k - 1 downsweeps
    {
        Scan_PassInput<T> *pass_input = input->pass_inputs + pass;
        int grid_dim = (pass_input->element_count + block_dim - 1)/block_dim;
        CUDACheck((ScanDownsweep<T><<<grid_dim, block_dim, 0, stream>>>(pass_input->element_count, pass_input->d_array, pass_input->d_summary)));
    }
}

#include "benchmarking.cu"

using InputType = int;

template <typename T>
struct Data
{
    u64 count;
    T *h_array;
    T *d_array;
    Scan_Input<T> *input;
};

template <typename T>
static Data<T> *CreateData(Arena *arena, u64 count, hipStream_t stream)
{
    Data<T> *data = PushStruct(arena, Data<T>);
    data->count = count;

    data->h_array = PushArray(arena, T, data->count);
    for (u64 i = 0; i < data->count; ++i)
        data->h_array[i] = T(1);

    CUDACheck(hipMalloc(&data->d_array, data->count*sizeof(*data->d_array)));
    CUDACheck(hipMemcpyAsync(data->d_array, data->h_array, data->count*sizeof(*data->d_array), hipMemcpyHostToDevice, stream));

    CUDACheck(hipStreamSynchronize(stream));

    data->input = Scan_CreateInput(arena, data->count, data->d_array, SEGMENT_SIZE);

    return data;
}

template <typename T>
static void DestroyData(Data<T> *data)
{
    Scan_DestroyInput(data->input);
    CUDACheck(hipFree(data->d_array));
}

template <typename T>
static b32 ValidateGPUOutput(Arena *arena, Data<T> *data)
{
    Scratch scratch = ScratchBegin(arena);

    T *gpu_out = PushArray(scratch.arena, T, data->count);
    CUDACheck(hipMemcpy(gpu_out, data->d_array, data->count*sizeof(*gpu_out), hipMemcpyDeviceToHost));

    SequentialScan<T>(data->count, data->h_array);
    b32 result = (memcmp(gpu_out, data->h_array, data->count*sizeof(*gpu_out)) == 0);

    ScratchEnd(&scratch);

    return result;
}

template <typename T>
static void FunctionToBenchmark(Data<T> *data, hipStream_t stream)
{
    Scan1<T>(data->input, stream);
}

static void TestScan();

int main(int argc, char **argv)
{
    char *file_name = 0;
    if (argc == 2)
    {
        file_name = argv[1];
    }

    if (1)
    {
        TestScan();
        printf("All tests passed\n");
    }

    f64 peak_gbps = 0.0;
    f64 peak_gflops = 0.0;
    GetPeakMeasurements(&peak_gbps, &peak_gflops, true);
    
    printf("Bandwidth (Peak):\t%.2f GBPS\n", peak_gbps);
    printf("Throughput (Peak):\t%.2f GFLOPS\n", peak_gflops);
    printf("Arithmetic intensity (Peak):\t%.2f FLOPS/byte\n", peak_gflops/peak_gbps);

    Benchmark<InputType>(peak_gbps, peak_gflops, file_name);

    return 0;
}

static void TestScan()
{
    srand(0);

    int test_count = 10;
    while (test_count > 0)
    {
        Scratch scratch = ScratchBegin(GetScratchArena(GigaBytes(10)));
        
        u64 count = GetRandomNumber(30);
        printf("Count: %llu", count);

        Data<int> *data = CreateData<int>(scratch.arena, count, 0);

        Scan_Input<int> *input = Scan_CreateInput(scratch.arena, count, data->d_array, SEGMENT_SIZE);
        Scan1<int>(input, 0);

        int *gpu_output = PushArray(scratch.arena, int, count);
        CUDACheck(hipMemcpy(gpu_output, data->d_array, count*sizeof(int), hipMemcpyDeviceToHost));

        SequentialScan<int>(count, data->h_array);
        for (int i = 0; i < count; ++i)
        {
            if (data->h_array[i] != gpu_output[i])
            {
                printf("[FAIL] Result[%d] (GPU): %d \tExpected: %d\n", i, gpu_output[i], data->h_array[i]);
                exit(1);
            }
        }
        printf(",\tPassed\n");

        DestroyData<int>(data);
        ScratchEnd(&scratch);

        --test_count;
    }
}